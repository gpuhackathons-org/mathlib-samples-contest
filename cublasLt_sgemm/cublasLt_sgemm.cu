#include "hip/hip_runtime.h"
/* Includes, system */
#include <cstdio>

/* Includes, cuda & thrust */
#include <hipblaslt.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

// *************** FOR ERROR CHECKING *******************
#ifndef CUDA_RT_CALL
#define CUDA_RT_CALL(call)                                                                         \
    do {                                                                                           \
        hipError_t err_ = (call);                                                                 \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA RT Error %d at %s:%d\n", err_, __FILE__, __LINE__);                  \
            throw std::runtime_error("CUDA RT Error");                                             \
        }                                                                                          \
    } while (0)
#endif // CUDA_RT_CALL

#ifndef CUBLAS_CALL
#define CUBLAS_CALL(call)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (call);                                                              \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("CUBLAS Error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
            throw std::runtime_error("CUBLAS Error");                                              \
        }                                                                                          \
    } while (0)
#endif // CUBLAS_CALL
// *************** FOR ERROR CHECKING *******************

#define ROW_MAJOR 1

/* Host implementation of a simple version of sgemm */
void simple_sgemm(int const &m, int const &n, int const &k, float const &alpha, float const *A,
                  float const *B, float const &beta, float *C) {

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float prod = 0;

            for (int l = 0; l < k; ++l) {
#if ROW_MAJOR
                prod += A[l + i * k] * B[j + l * n];
#else
                prod += A[l * m + i] * B[j * k + l];
#endif
            }
#if ROW_MAJOR
            C[j + i * m] = alpha * prod + beta * C[j + i * m];
#else
            C[j * m + i] = alpha * prod + beta * C[j * m + i];
#endif
        }
    }
}

void LtSgemm(hipblasLtHandle_t ltHandle, hipblasOperation_t transa, hipblasOperation_t transb,
             int const &m, int const &n, int const &k, float const *alpha, float const *A,
             int const &lda, float const *B, int const &ldb, float const *beta, float *C,
             int const &ldc, void *workspace, size_t workspaceSize) {

    hipblasLtMatmulDesc_t operationDesc = nullptr;
    hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr;
    hipblasLtMatmulPreference_t preference = nullptr;

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

#if ROW_MAJOR
    hipblasLtOrder_t rowOrder = HIPBLASLT_ORDER_ROW;
#endif

    // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
    // for details about defaults; here we just set the transforms for
    // A and B.
    CUBLAS_CALL(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    CUBLAS_CALL(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa,
                                               sizeof(transa)));
    CUBLAS_CALL(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb,
                                               sizeof(transa)));

    // Create matrix descriptors. Not setting any extra attributes.
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k,
                                           transa == HIPBLAS_OP_N ? k : m, lda));
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n,
                                           transb == HIPBLAS_OP_N ? n : k, ldb));
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

#if ROW_MAJOR
    CUBLAS_CALL(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &rowOrder,
                                                 sizeof(rowOrder)));
    CUBLAS_CALL(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &rowOrder,
                                                 sizeof(rowOrder)));
    CUBLAS_CALL(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &rowOrder,
                                                 sizeof(rowOrder)));
#endif

    // Create preference handle; In general, extra attributes can be
    // used here to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C. However, for simplicity
    // here we assume A,B,C are always well aligned (e.g., directly
    // come from hipMalloc)
    CUBLAS_CALL(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CALL(hipblasLtMatmulPreferenceSetAttribute(preference,
                                                     HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                                     &workspaceSize, sizeof(workspaceSize)));

    // We just need the best available heuristic to try and run matmul.
    // There is no guarantee that this will work. For example, if A is
    // badly aligned, you can request more (e.g. 32) algos and try to
    // run them one by one until something works.
    CUBLAS_CALL(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc,
                                               preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0)
        throw std::runtime_error("!!!! Unable to find any suitable algorithms");

    CUBLAS_CALL(hipblasLtMatmul(ltHandle, operationDesc, alpha, A, Adesc, B, Bdesc, beta, C, Cdesc,
                               C, Cdesc, &heuristicResult.algo, workspace, workspaceSize, 0));

    // Descriptors are no longer needed as all GPU work was already
    // enqueued.
    CUBLAS_CALL(hipblasLtMatmulPreferenceDestroy(preference));
    CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Cdesc));
    CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Bdesc));
    CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Adesc));
    CUBLAS_CALL(hipblasLtMatmulDescDestroy(operationDesc));
}

void calculate(int const &m, int const &n, int const &k) {

    float alpha = 1.0f, beta = 0.0f;
    int lda = k, ldb = n, ldc = n;
    void *d_workspace = nullptr;

    size_t sizeA = m * k;
    size_t sizeB = k * n;
    size_t sizeC = m * n;
    size_t workspaceSize = 4096;

    float error_norm = 0.0f;
    float ref_norm = 0.0f;
    float diff = 0.0f;

    hipblasLtHandle_t handle;

    /* Initialize cuBLASLt */
    CUBLAS_CALL(hipblasLtCreate(&handle));

    /* Allocate device memory for workspace */
    CUDA_RT_CALL(hipMalloc((void **)&d_workspace, workspaceSize));

    /* Initialize CUBLAS */
    printf("cuBLASLt %dx%dx%d test running..\n", m, n, k);

    /* Allocate host memory for the matrices */
    thrust::host_vector<float> h_A(sizeA, 0.0f);
    thrust::host_vector<float> h_B(sizeB, 0.0f);
    thrust::host_vector<float> h_C(sizeC, 0.0f);
    thrust::host_vector<float> h_C_ref(sizeC, 0.0f);

    /* Fill the matrices with test data */
    /* Assume square matrices */
    for (int i = 0; i < m * m; i++) {
        h_A[i] = rand() / static_cast<float>(RAND_MAX);
        h_B[i] = rand() / static_cast<float>(RAND_MAX);
    }

    /* Allocate device memory for the matrices */
    thrust::device_vector<float> d_A(h_A);
    thrust::device_vector<float> d_B(h_B);
    thrust::device_vector<float> d_C(sizeC, 0.0f);

    /* Retrieve raw pointer for device data */
    float *d_A_ptr = thrust::raw_pointer_cast(&d_A[0]);
    float *d_B_ptr = thrust::raw_pointer_cast(&d_B[0]);
    float *d_C_ptr = thrust::raw_pointer_cast(&d_C[0]);

    /* Performs operation using plain C code */
    simple_sgemm(m, n, k, alpha, h_A.data(), h_B.data(), beta, h_C_ref.data());

    LtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A_ptr, lda, d_B_ptr, ldb, &beta,
            d_C_ptr, ldc, d_workspace, workspaceSize);

    CUDA_RT_CALL(hipDeviceSynchronize());

    /* Allocate host memory for reading back the result from device memory */
    h_C = d_C;

    /* Check result against reference */
    for (int i = 0; i < m * m; i++) {
        diff = h_C_ref[i] - h_C[i];
        error_norm += diff * diff;
        ref_norm += h_C_ref[i] * h_C_ref[i];
    }

    error_norm = static_cast<float>(sqrt(static_cast<double>(error_norm)));
    ref_norm = static_cast<float>(sqrt(static_cast<double>(ref_norm)));

    if (fabs(ref_norm) < 1e-7)
        throw std::runtime_error("!!!! reference norm is 0\n");

    /* Shutdown */
    CUBLAS_CALL(hipblasLtDestroy(handle));

    if (error_norm / ref_norm < 1e-4f)
        printf("cuBLASLt SGEMM test passed.\n");
    else
        throw std::runtime_error("!!!! cuBLASLt SGEMM test failed.\n");
}

/* Main */
int main(int argc, char **argv) {

    // Compute square matrices
    calculate(1024, 1024, 1024);

    return (EXIT_SUCCESS);
}